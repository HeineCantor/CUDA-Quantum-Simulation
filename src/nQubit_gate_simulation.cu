#include "hip/hip_runtime.h"
#include "../include/nQubit_gate_simulation.cuh"

__global__ void LSB_nQubit_kernel(hipDoubleComplex* stateVector)
{
    __shared__ hipDoubleComplex subCoefficients[1 << MAX_QUBITS_PER_SM];

    int threadIndex = threadIdx.x;
    int kIndex = blockIdx.x / (MAX_QUBITS_PER_SM + blockIdx.x + 1);    // blockIndex -> k coefficient

    if(threadIndex < twoToThePower(MAX_QUBITS_PER_SM))
        subCoefficients[threadIndex] = stateVector[kIndex ^ threadIndex];

    for(int i = 0; i < MAX_QUBITS_PER_SM; i++)
    {
        __syncthreads();

        if(threadIndex < twoToThePower(MAX_QUBITS_PER_SM - 1))
        {
            int xorOffset = (1 << i); //2^qubit_index

            int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
            int iXORCoeff = iCoeff ^ xorOffset;

            hipDoubleComplex coefficients[2] = {subCoefficients[iCoeff], subCoefficients[iXORCoeff]};

            gates::gate_hadamard(coefficients);

            subCoefficients[iCoeff] = coefficients[0];
            subCoefficients[iXORCoeff] = coefficients[1];
        }
    }

    __syncthreads();

    if(threadIndex < twoToThePower(MAX_QUBITS_PER_SM))
        stateVector[kIndex ^ threadIndex] = subCoefficients[threadIndex];
}

__global__ void MSB_nQubit_kernel(hipDoubleComplex* stateVector, int startingQubit)
{
    __shared__ hipDoubleComplex subCoefficients[1 << MAX_QUBITS_PER_SM];

    int threadIndex = threadIdx.x;
    int kIndex = blockIdx.x;    // blockIndex -> k coefficient

    int twoToTheQ = twoToThePower(startingQubit);

    if(threadIndex < twoToThePower(MAX_QUBITS_PER_SM))
        subCoefficients[threadIndex] = stateVector[kIndex ^ (twoToTheQ * threadIndex)];
 
    for(int i = 0; i < MAX_QUBITS_PER_SM; i++)
    {
        __syncthreads();

        if(threadIndex < twoToThePower(MAX_QUBITS_PER_SM - 1))
        {
            int xorOffset = (1 << i); //2^qubit_index

            int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
            int iXORCoeff = iCoeff ^ xorOffset;

            hipDoubleComplex coefficients[2] = {subCoefficients[iCoeff], subCoefficients[iXORCoeff]};

            gates::gate_hadamard(coefficients);

            subCoefficients[iCoeff] = coefficients[0];
            subCoefficients[iXORCoeff] = coefficients[1];
        }
    }

    __syncthreads();

    if(threadIndex < twoToThePower(MAX_QUBITS_PER_SM))
        stateVector[kIndex ^ (twoToTheQ * threadIndex)] = subCoefficients[threadIndex];
}

void nQubitGateSimulation()
{
    int statesNumber = twoToThePower(NUM_QUBITS);
    int stateVectorSize = sizeof(hipDoubleComplex) * statesNumber;

    int blockNumber = twoToThePower(NUM_QUBITS - MAX_QUBITS_PER_SM);

    printNQubitsSimulationDetails(NUM_QUBITS, blockNumber);

    hipDoubleComplex unitaryComplex;
    unitaryComplex.x = 1;
    unitaryComplex.y = 0;

    hipDoubleComplex* hostStateVector = new hipDoubleComplex[stateVectorSize];

    hipDoubleComplex* deviceStateVector = NULL;

    CHKERR( hipMalloc((void**)& deviceStateVector, stateVectorSize) );

    // Initializing the state vector with the state |000...0>, a.k.a. the state vector [ 1 0 0 ... 0 ]
    CHKERR( hipMemset(deviceStateVector, 0, stateVectorSize) );
    CHKERR( hipMemcpy(deviceStateVector, &unitaryComplex, sizeof(hipDoubleComplex), hipMemcpyHostToDevice) ); 

    // LSB Kernel Call
    LSB_nQubit_kernel<<<blockNumber, THREAD_PER_BLOCK>>>(deviceStateVector);

    CHKERR( hipPeekAtLastError() );

    // MSB Kernel Call
    MSB_nQubit_kernel<<<blockNumber, THREAD_PER_BLOCK>>>(deviceStateVector, NUM_QUBITS / 2);

    CHKERR( hipPeekAtLastError() );

    CHKERR( hipMemcpy(hostStateVector, deviceStateVector, stateVectorSize, hipMemcpyDeviceToHost) );
    CHKERR( hipFree(deviceStateVector) );

    //printStateVector(hostStateVector, statesNumber);
    printQubitsState(hostStateVector, NUM_QUBITS);

    free(hostStateVector);
}