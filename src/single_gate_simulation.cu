#include "hip/hip_runtime.h"
#include "../include/single_gate_simulation.cuh"

__global__ void single_X_kernel(hipDoubleComplex* stateVector, int statesNumber, int qubit_index)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex < statesNumber / 2)
    {
        int xorOffset = (1 << qubit_index); //2^qubit_index

        int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
        int iXORCoeff = iCoeff ^ xorOffset;

        hipDoubleComplex coefficients[2] = {stateVector[iCoeff], stateVector[iXORCoeff]};

        gates::gate_x(coefficients);

        stateVector[iCoeff] = coefficients[0];
        stateVector[iXORCoeff] = coefficients[1];
    }
}

__global__ void single_CNOT_kernel(hipDoubleComplex* stateVector, int statesNumber, int qubit_controlled, int qubit_controller)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex < statesNumber / 4)
    {
        int xorOffset = twoToThePower(qubit_controlled);

        int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
        int iXORCoeff = iCoeff ^ xorOffset;

        hipDoubleComplex coefficients[2] = {stateVector[iCoeff], stateVector[iXORCoeff]};

        gates::gate_x(coefficients);

        stateVector[iCoeff] = coefficients[0];
        stateVector[iXORCoeff] = coefficients[1];
    }
}

__global__ void single_Z_kernel(hipDoubleComplex* stateVector, int statesNumber, int qubit_index)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex < statesNumber / 2)
    {
        int xorOffset = (1 << qubit_index); //2^qubit_index

        int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
        int iXORCoeff = iCoeff ^ xorOffset;

        hipDoubleComplex coefficients[2] = {stateVector[iCoeff], stateVector[iXORCoeff]};

        gates::gate_z(coefficients);

        stateVector[iCoeff] = coefficients[0];
        stateVector[iXORCoeff] = coefficients[1];
    }
}

__global__ void single_hadamard_kernel(hipDoubleComplex* stateVector, int statesNumber, int qubit_index)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex < statesNumber / 2)
    {
        int xorOffset = (1 << qubit_index); //2^qubit_index

        int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
        int iXORCoeff = iCoeff ^ xorOffset;

        hipDoubleComplex coefficients[2] = {stateVector[iCoeff], stateVector[iXORCoeff]};

        gates::gate_hadamard(coefficients);

        stateVector[iCoeff] = coefficients[0];
        stateVector[iXORCoeff] = coefficients[1];
    }
}

void singleGateSimulation()
{
    int statesNumber = twoToThePower(NUM_QUBITS);
    int stateVectorSize = sizeof(hipDoubleComplex) * statesNumber;

    int requiredThreads = statesNumber / 2;
    int blockNumber = (requiredThreads + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

    printSingleQubitSimulationDetails(NUM_QUBITS, requiredThreads, blockNumber);

    hipDoubleComplex unitaryComplex;
    unitaryComplex.x = 1;
    unitaryComplex.y = 0;

    hipDoubleComplex* hostStateVector = new hipDoubleComplex[stateVectorSize];

    hipDoubleComplex* deviceStateVector = NULL;

    hipEvent_t start, stop;
    float mainStreamElapsedTime;

    CHKERR( hipEventCreate(&start) );
    CHKERR( hipEventCreate(&stop) );

    CHKERR( hipEventRecord( start, 0 ) );

    CHKERR( hipMalloc((void**)& deviceStateVector, stateVectorSize) );

    // Initializing the state vector with the state |000...0>, a.k.a. the state vector [ 1 0 0 ... 0 ]
    CHKERR( hipMemset(deviceStateVector, 0, stateVectorSize) );
    CHKERR( hipMemcpy(deviceStateVector, &unitaryComplex, sizeof(hipDoubleComplex), hipMemcpyHostToDevice) ); 

    //single_X_kernel<<<blockNumber, THREAD_PER_BLOCK>>>(deviceStateVector, statesNumber, 0);

    //single_Z_kernel<<<blockNumber, THREAD_PER_BLOCK>>>(deviceStateVector, statesNumber, 0);

    for(int i = 0; i < NUM_QUBITS; i++)
        single_hadamard_kernel<<<blockNumber, THREAD_PER_BLOCK>>>(deviceStateVector, statesNumber, i);

    CHKERR( hipPeekAtLastError() ); 

    CHKERR( hipMemcpy(hostStateVector, deviceStateVector, stateVectorSize, hipMemcpyDeviceToHost) );
    
    CHKERR( hipEventRecord( stop, 0 ) );

	CHKERR( hipEventSynchronize( stop ) );
	CHKERR( hipEventElapsedTime( &mainStreamElapsedTime, start, stop ) );
	CHKERR( hipEventDestroy( start ) );
	CHKERR( hipEventDestroy( stop ) );

    CHKERR( hipFree(deviceStateVector) );

    //printStateVector(hostStateVector, statesNumber, 4);

    cout << "Simulation elapsed time: " << mainStreamElapsedTime <<  " ms." << endl;

    free(hostStateVector);
}