#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>

#include "error.cuh"
#include "gate.cuh"
#include "print_util.h"

#define NUM_QUBITS 20
#define THREAD_PER_BLOCK 256

__host__ __device__ inline int twoToThePower(int exp)
{
    return 1 << exp;
}

__global__ void single_X_kernel(hipDoubleComplex* stateVector, int statesNumber, int qubit_index)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex < statesNumber)
    {
        int xorOffset = (1 << qubit_index); //2^qubit_index

        int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
        int iXORCoeff = iCoeff ^ xorOffset;

        hipDoubleComplex coefficients[2] = {stateVector[iCoeff], stateVector[iXORCoeff]};

        gates::gate_x(coefficients);

        stateVector[iCoeff] = coefficients[0];
        stateVector[iXORCoeff] = coefficients[1];
    }
}

__global__ void single_Z_kernel(hipDoubleComplex* stateVector, int statesNumber, int qubit_index)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex < statesNumber)
    {
        int xorOffset = (1 << qubit_index); //2^qubit_index

        int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
        int iXORCoeff = iCoeff ^ xorOffset;

        hipDoubleComplex coefficients[2] = {stateVector[iCoeff], stateVector[iXORCoeff]};

        gates::gate_z(coefficients);

        stateVector[iCoeff] = coefficients[0];
        stateVector[iXORCoeff] = coefficients[1];
    }
}

__global__ void single_hadamard_kernel(hipDoubleComplex* stateVector, int statesNumber, int qubit_index)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex < statesNumber)
    {
        int xorOffset = (1 << qubit_index); //2^qubit_index

        int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
        int iXORCoeff = iCoeff ^ xorOffset;

        hipDoubleComplex coefficients[2] = {stateVector[iCoeff], stateVector[iXORCoeff]};

        gates::gate_hadamard(coefficients);

        stateVector[iCoeff] = coefficients[0];
        stateVector[iXORCoeff] = coefficients[1];
    }
}

int main()
{
    int statesNumber = twoToThePower(NUM_QUBITS);
    int stateVectorSize = sizeof(hipDoubleComplex) * statesNumber;

    int requiredThreads = statesNumber / 2;
    int blockNumber = (requiredThreads + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

    printSimulationDetails(NUM_QUBITS, requiredThreads, blockNumber);

    hipDoubleComplex unitaryComplex;
    unitaryComplex.x = 1;
    unitaryComplex.y = 0;

    hipDoubleComplex* hostStateVector = new hipDoubleComplex[stateVectorSize];

    hipDoubleComplex* deviceStateVector = NULL;

    CHKERR( hipMalloc((void**)& deviceStateVector, stateVectorSize) );

    // Initializing the state vector with the state |000...0>, a.k.a. the state vector [ 1 0 0 ... 0 ]
    CHKERR( hipMemset(deviceStateVector, 0, stateVectorSize) );
    CHKERR( hipMemcpy(deviceStateVector, &unitaryComplex, sizeof(hipDoubleComplex), hipMemcpyHostToDevice) ); 

    single_X_kernel<<<blockNumber, THREAD_PER_BLOCK>>>(deviceStateVector, statesNumber, 0);

    single_Z_kernel<<<blockNumber, THREAD_PER_BLOCK>>>(deviceStateVector, statesNumber, 0);

    single_hadamard_kernel<<<blockNumber, THREAD_PER_BLOCK>>>(deviceStateVector, statesNumber, 0);
    single_hadamard_kernel<<<blockNumber, THREAD_PER_BLOCK>>>(deviceStateVector, statesNumber, 1);

    CHKERR( hipPeekAtLastError() ); 

    CHKERR( hipMemcpy(hostStateVector, deviceStateVector, stateVectorSize, hipMemcpyDeviceToHost) );
    CHKERR( hipFree(deviceStateVector) );

    printStateVector(hostStateVector, statesNumber, 4);

    free(hostStateVector);

    return 0;
}