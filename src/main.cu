#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <iostream>
#include "error.cuh"
#include "gate.cuh"

#define NUM_QUBITS 2

#define THREAD_PER_BLOCK 256

using namespace std;

__device__ inline hipDoubleComplex* gate_x(hipDoubleComplex* inputCoefficients)
{
    hipDoubleComplex outputCoefficients[2];

    outputCoefficients[0] = inputCoefficients[1];
    outputCoefficients[1] = inputCoefficients[0];

    return outputCoefficients;
}

__device__ inline hipDoubleComplex* gate_z(hipDoubleComplex* inputCoefficients)
{
    hipDoubleComplex outputCoefficients[2];

    hipDoubleComplex minusOne;
    minusOne.x = -1;
    minusOne.y = 0;

    outputCoefficients[0] = inputCoefficients[0];
    outputCoefficients[1] = hipCmul(minusOne, inputCoefficients[1]);

    return outputCoefficients;
}

__device__ inline hipDoubleComplex* gate_hadamard(hipDoubleComplex* inputCoefficients)
{
    hipDoubleComplex outputCoefficients[2];

    hipDoubleComplex sqrt2;
    sqrt2.x = 0.7071067811865475;
    sqrt2.y = 0;

    hipDoubleComplex minusSqrt2;
    minusSqrt2.x = -0.7071067811865475;
    minusSqrt2.y = 0;


    outputCoefficients[0] = hipCadd(hipCmul(sqrt2, inputCoefficients[0]), hipCmul(sqrt2, inputCoefficients[1]));
    outputCoefficients[1] = hipCadd(hipCmul(sqrt2, inputCoefficients[0]), hipCmul(minusSqrt2, inputCoefficients[1]));

    return outputCoefficients;
}

inline int twoToThePower(int exp)
{
    return 1 << exp;
}

__global__ void single_X_kernel(hipDoubleComplex* stateVector, int vectorCount, int qubit_index)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    int xorOffset = (1 << qubit_index); //2^qubit_index

    int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
    int iXORCoeff = iCoeff ^ xorOffset;

    hipDoubleComplex* coefficients = new hipDoubleComplex[2]();
    coefficients[0] = stateVector[iCoeff];
    coefficients[1] = stateVector[iXORCoeff];

    coefficients = gate_x(coefficients);

    stateVector[iCoeff] = coefficients[0];
    stateVector[iXORCoeff] = coefficients[1];
}

__global__ void single_Z_kernel(hipDoubleComplex* stateVector, int vectorCount, int qubit_index)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    int xorOffset = (1 << qubit_index); //2^qubit_index

    int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
    int iXORCoeff = iCoeff ^ xorOffset;

    hipDoubleComplex* coefficients = new hipDoubleComplex[2]();
    coefficients[0] = stateVector[iCoeff];
    coefficients[1] = stateVector[iXORCoeff];

    coefficients = gate_z(coefficients);

    stateVector[iCoeff] = coefficients[0];
    stateVector[iXORCoeff] = coefficients[1];
}

__global__ void single_hadamard_kernel(hipDoubleComplex* stateVector, int vectorCount, int qubit_index)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    int xorOffset = (1 << qubit_index); //2^qubit_index

    int iCoeff = threadIndex + (threadIndex / xorOffset) * xorOffset;
    int iXORCoeff = iCoeff ^ xorOffset;

    hipDoubleComplex* coefficients = new hipDoubleComplex[2]();
    coefficients[0] = stateVector[iCoeff];
    coefficients[1] = stateVector[iXORCoeff];

    coefficients = gate_hadamard(coefficients);

    stateVector[iCoeff] = coefficients[0];
    stateVector[iXORCoeff] = coefficients[1];
}

void printStateVector(hipDoubleComplex* vector, int vectorCount)
{
    cout << "Output State Vector: [ ";
    for(int i = 0; i < vectorCount; i++)
    {
        cout << "(" << vector[i].x << " + ";
        cout << vector[i].y << "i)";

        if(i < vectorCount - 1)
            cout << ", ";
    }

    cout << " ]" << endl;
}

int main()
{
    int statesNumber = twoToThePower(NUM_QUBITS);
    int stateVectorSize = sizeof(hipDoubleComplex) * statesNumber;

    int threadNumber = statesNumber / 2;
    int blockNumber = (threadNumber + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

    cout << "Qubit number: " << NUM_QUBITS << endl;
    cout << "States number: " << twoToThePower(NUM_QUBITS) << endl;

    cout << endl;

    cout << "======= SIMULATION FOR ONE QUBIT GATE =======" << endl;
    cout << "Required Thread number: " << threadNumber << endl;
    cout << "Required Blocks number: " << blockNumber << endl;

    cout << endl;

    hipDoubleComplex unitaryComplex;
    unitaryComplex.x = 1;
    unitaryComplex.y = 0;

    hipDoubleComplex* hostStateVector = new hipDoubleComplex[stateVectorSize];

    hipDoubleComplex* deviceStateVector = NULL;

    CHKERR( hipMalloc((void**)& deviceStateVector, stateVectorSize) );

    // Initializing the state vector with the state |000...0>, a.k.a. the state vector [ 1 0 0 ... 0 ]
    CHKERR( hipMemset(deviceStateVector, 0, stateVectorSize) );
    CHKERR( hipMemcpy(deviceStateVector, &unitaryComplex, sizeof(hipDoubleComplex), hipMemcpyHostToDevice) ); 

    //single_X_kernel<<<blockNumber, threadNumber>>>(deviceStateVector, statesNumber, 0);

    //single_Z_kernel<<<blockNumber, threadNumber>>>(deviceStateVector, statesNumber, 0);

    single_hadamard_kernel<<<blockNumber, threadNumber>>>(deviceStateVector, statesNumber, 0);
    single_hadamard_kernel<<<blockNumber, threadNumber>>>(deviceStateVector, statesNumber, 1);

    CHKERR( hipMemcpy(hostStateVector, deviceStateVector, stateVectorSize, hipMemcpyDeviceToHost) );
    CHKERR( hipFree(deviceStateVector) );

    printStateVector(hostStateVector, statesNumber);

    free(hostStateVector);

    return 0;
}