#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <iostream>
#include "error.cuh"
#include "gate.cuh"

#define NUM_QUBITS 2

#define THREAD_PER_BLOCK 256

using namespace std;

__device__ inline hipComplex gate_x(hipComplex a, hipComplex b)
{
    return b;
}

inline int twoToThePower(int exp)
{
    return 1 << exp;
}

__global__ void single_X_kernel(hipComplex* stateVector, int vectorCount, int qubit_index)
{
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    hipComplex currentCoefficient = stateVector[threadIndex];
    hipComplex xorCoefficient = stateVector[threadIndex ^ (1 << qubit_index)];

    stateVector[threadIndex] = gate_x(currentCoefficient, xorCoefficient);
    stateVector[threadIndex ^ (1 << qubit_index)] = gate_x(xorCoefficient, currentCoefficient);
}

void printStateVector(hipComplex* vector, int vectorCount)
{
    cout << "Output State Vector: [ ";
    for(int i = 0; i < vectorCount; i++)
    {
        cout << vector[i].x << ".";
        cout << vector[i].y;

        if(i < vectorCount - 1)
            cout << ", ";
    }

    cout << " ]" << endl;
}

int main()
{
    int statesNumber = twoToThePower(NUM_QUBITS);
    int stateVectorSize = sizeof(hipComplex) * statesNumber;

    int threadNumber = statesNumber / 2;
    int blockNumber = (threadNumber + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

    cout << "Qubit number: " << NUM_QUBITS << endl;
    cout << "States number: " << twoToThePower(NUM_QUBITS) << endl;

    cout << endl;

    cout << "======= SIMULATION FOR ONE QUBIT GATE =======" << endl;
    cout << "Required Thread number: " << threadNumber << endl;
    cout << "Required Blocks number: " << blockNumber << endl;

    cout << endl;

    hipComplex unitaryComplex;
    unitaryComplex.x = 1;
    unitaryComplex.y = 0;

    hipComplex* hostStateVector = new hipComplex[stateVectorSize];

    hipComplex* deviceStateVector = NULL;

    CHKERR( hipMalloc((void**)& deviceStateVector, stateVectorSize) );

    // Initializing the state vector with the state |000...0>, a.k.a. the state vector [ 1 0 0 ... 0 ]
    CHKERR( hipMemset(deviceStateVector, 0, stateVectorSize) );
    CHKERR( hipMemcpy(deviceStateVector, &unitaryComplex, sizeof(hipComplex), hipMemcpyHostToDevice) ); 

    single_X_kernel<<<blockNumber, threadNumber>>>(deviceStateVector, statesNumber, 1);

    CHKERR( hipMemcpy(hostStateVector, deviceStateVector, stateVectorSize, hipMemcpyDeviceToHost) );
    CHKERR( hipFree(deviceStateVector) );

    printStateVector(hostStateVector, statesNumber);

    free(hostStateVector);

    return 0;
}